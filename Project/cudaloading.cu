#include"cudaloading.cuh"
#include ""
#include "hip/hip_runtime.h"

cudaloading::cudaloading(const readmat& MAT,int alpha)
{
	thisData.elements = MAT.getarraypointer();
	dimensionpointer = MAT.dimensionpointer();
	thisData.X = *(dimensionpointer);
	thisData.Y = *(dimensionpointer + 1);
	thisData.U = *(dimensionpointer + 2);
	thisData.V = *(dimensionpointer + 3);
	thisData.alpha = alpha;



}

void cudaloading::cudacopy()
{

}


