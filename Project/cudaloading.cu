#include"cudaloading.cuh"


cudaloading::cudaloading(const readmat& MAT,int alpha)
{
	
	dimensionpointer = MAT.dimensionpointer();
	thisData.X = *(dimensionpointer);
	thisData.Y = *(dimensionpointer + 1);
	thisData.U = *(dimensionpointer + 2);
	thisData.V = *(dimensionpointer + 3);
	thisData.alpha = alpha;
	// allocation of memory in device for data
	size_of_elements = thisData.X*thisData.Y*thisData.U*thisData.V;
	
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		throw device_operation_exception("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	
	cudaStatus = hipMalloc((void**)&thisData.elements, size_of_elements * sizeof(double));
	if (cudaStatus != hipSuccess) {
		throw device_operation_exception("Device Memory allocation failed");
	}

	cudaStatus = hipMemcpy(thisData.elements, MAT.getarraypointer(), size_of_elements * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		throw device_operation_exception("Device Memory copy to device failed");
	}



}

void cudaloading::cudacopy()
{

}


