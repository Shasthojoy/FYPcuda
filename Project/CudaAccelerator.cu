#include"CudaAccelerator.cuh"


CudaAccelerator::CudaAccelerator(Data_Struct_In ImageMetaData, float Alpha)
{

		// Aquiring LightField Meta data and Other information
		size_of_elements = ImageMetaData.X*ImageMetaData.Y*ImageMetaData.U*ImageMetaData.V;
		AlphaValue = Alpha;
		X = ImageMetaData.X;
		Y = ImageMetaData.Y;
		U = ImageMetaData.U;
		V = ImageMetaData.V;
		Exception_Device.throw_cuda_error(hipSetDevice(0));
		//Allocate memory In device
		Exception_Device.throw_cuda_error(hipMalloc((void**)&Device_Input, size_of_elements * sizeof(float)));
		// Allocate host pinned memory
		Exception_Device.throw_cuda_error(hipHostMalloc((void**)&HostPinnedImage, size_of_elements * sizeof(float)));


}

void CudaAccelerator::FractionalShift()
{

}


